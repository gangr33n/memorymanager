#include "VariableSizeMemoryManager.h"

/**
 * Constructor for the variable length Memory Manager object, assigns total memory
 * frame
 *
 * @param numBuckets Number of items to be stored
 * @param maxItemSize Maximum iize to be used for each item by the Memory
 * Manager
 */
VariableSizeMemoryManager::VariableSizeMemoryManager(unsigned int dimension,
                                                      unsigned int maxItemSize)
{
   unsigned int i;

   setItemSize(maxItemSize);
   setDimension(dimension);

   bitmap = new Bitmap(getNumBuckets()*getItemSize());
   references = new VariableSizeBucket[getNumBuckets()];
   if (bitmap == NULL || references == NULL || setMemory() == FAILURE)
   {
      std::cout << "Unable to allocate memory for manager! Exiting...\n";
      exit(EXIT_FAILURE);
   }

   for (i = 0; i < getNumBuckets(); i++)
   {
      references[i].size = 0;
	  references[i].firstByte = NULL;
   }

   nextFree = 0;
}

/**
 * Deconstructor for the fixed length Memory Manager object, frees the total
 * memory frame
 */
VariableSizeMemoryManager::~VariableSizeMemoryManager()
{
   cleanup();
}

/**
 * Returns a pointer to the bucket specified
 *
 * @return Result of operation, SUCCESS if successful, FAILURE if unsuccessful.
 * Process is unsuccessful when the bucket is unused
 * @param dest Location to store retreived data in
 * @param x The x coordinate of the cell to be retrieved
 * @param y The y coordinate of the cell to be retrieved
 */
unsigned int VariableSizeMemoryManager::get(void* dest, unsigned int x,
                                                               unsigned int y)
{
   if (references[x*getDimension()+y].firstByte == NULL)
      return FAILURE;
   hipMemcpy(dest, references[x*getDimension()+y].firstByte, getItemSize(), hipMemcpyDeviceToHost);
   return SUCCESS;
}

/**
 * Sets the value of an already allocated bucket.
 *
 * @return Result of operation, SUCCESS if successful, FAILURE if unsuccessful.
 * Process is unsuccessful when the target is not already allocated
 * @param x The x coordinate of the cell to be retrieved
 * @param y The y coordinate of the cell to be retrieved
 * @param data Pointer to the data that is to be stored
 * @param size Size of the data to be stored
 */
unsigned int VariableSizeMemoryManager::set(unsigned int x, unsigned int y, void* data,
                                                            unsigned int size)
{
   unsigned int i, j, flag, temp;

   if (size > getItemSize())
      return FAILURE;

   for (i = nextFree; i != nextFree-1; i++)
   {
      flag = FALSE;
      for (j = 0; j < size; j++)
         if (bitmap->getBit(i+j) == USED)
         {
            flag = TRUE;
            break;
         }
      if (flag)
      {
         i += j;
         continue;
      }

      bitmap->setBits(i, size);
	  references[x*getDimension()+y].size = size;
	  references[x*getDimension()+y].firstByte = getMemory() + i;
	  hipMemcpy(references[x*getDimension()+y].firstByte, data, size, hipMemcpyHostToDevice);

      temp = nextFree;
      do
      {
         if (bitmap->getBit(nextFree) == EMPTY)
            break;
         nextFree++;
         if (nextFree == getNumBuckets())
            nextFree = 0;
      } while (temp != nextFree);

      return SUCCESS;
   }
   return FAILURE;
}

/**
 * Deletes the contents of a selected bucket
 *
 * @return Result of operation, SUCCESS if successful, FAILURE if unsuccessful.
 * Process is unsuccessful when the target is not already allocated
 * @param x The x coordinate of the cell to be retrieved
 * @param y The y coordinate of the cell to be retrieved
 */
unsigned int VariableSizeMemoryManager::del(unsigned int x, unsigned int y)
{
   unsigned int bit;

   if (references[x*getDimension()+y].firstByte == NULL)
      return FAILURE;
   
   bit = references[x*getDimension()+y].firstByte - getMemory();
   bitmap->clearBits(bit, references[x*getDimension()+y].size);

   references[x*getDimension()+y].firstByte = NULL;
   references[x*getDimension()+y].size = 0;
   
   return SUCCESS;
}

/**
 * Free any dynamic memory assigned to the manager
 */
void VariableSizeMemoryManager::cleanup()
{
   delete[] references;
   delete bitmap;
}
