#include "MemoryMonitor.h"

/**
 * Constructor for the variable length Memory Monitor object
 *
 * @param numBuckets Number of items to be stored
 * @param maxItemSize Maximum iize to be used for each item by the Memory
 * Manager 
 */
MemoryMonitor::MemoryMonitor(unsigned int dimension, unsigned int maxItemSize)
{
   unsigned int i;

   setItemSize(maxItemSize);
   setDimension(dimension);

   bitmap = new Bitmap(getNumBuckets()*getItemSize());
   references = new MonitorBucket[getNumBuckets()];
   if (bitmap == NULL || references == NULL || setMemory() == FAILURE)
   {
      std::cout << "Unable to allocate memory for manager! Exiting...\n";
      exit(EXIT_FAILURE);
   }

   for (i = 0; i < getNumBuckets(); i++)
   {
      references[i].size = 0;
	  references[i].firstByte = NULL;
   }

   nextFree = 0;
}

/**
 * Deconstructor for the Memory Monitor
 */
MemoryMonitor::~MemoryMonitor()
{
   cleanup();
}

/**
 * Returns a pointer to the bucket specified
 *
 * @return Result of operation, SUCCESS if successful, FAILURE if unsuccessful.
 * Process is unsuccessful when the bucket is unused
 * @param dest Location to store retreived data in
 * @param x The x coordinate of the cell to be retrieved
 * @param y The y coordinate of the cell to be retrieved
 */
unsigned int MemoryMonitor::get(void* dest, unsigned int x, unsigned int y)
{
   if (references[x*getDimension()+y].firstByte == NULL)
      return FAILURE;
   hipMemcpy(dest, references[x*getDimension()+y].firstByte, getItemSize(), hipMemcpyDeviceToHost);
   return SUCCESS;
}

/**
 * Sets the value of an already allocated bucket.
 *
 * @return Result of operation, SUCCESS if successful, FAILURE if unsuccessful.
 * Process is unsuccessful when the target is not already allocated
 * @param x The x coordinate of the cell to be retrieved
 * @param y The y coordinate of the cell to be retrieved
 * @param data Pointer to the data that is to be stored
 * @param size Size of the data to be stored
 */
unsigned int MemoryMonitor::set(unsigned int x, unsigned int y, void* data,
                                                            unsigned int size)
{
   unsigned int i, j, flag, temp;

   if (size > getItemSize())
      return FAILURE;

   for (i = nextFree; i != nextFree-1; i++)
   {
      flag = FALSE;
      for (j = 0; j < size; j++)
         if (bitmap->getBit(i+j) == USED)
         {
            flag = TRUE;
            break;
         }
      if (flag)
      {
         i += j;
         continue;
      }

      bitmap->setBits(i, size);
	  references[x*getDimension()+y].size = size;
	  references[x*getDimension()+y].firstByte = getMemory() + i;
	  hipMemcpy(references[x*getDimension()+y].firstByte, data, size, hipMemcpyHostToDevice);

      temp = nextFree;
      do
      {
         if (bitmap->getBit(nextFree) == EMPTY)
            break;
         nextFree++;
         if (nextFree == getNumBuckets())
            nextFree = 0;
      } while (temp != nextFree);

      return SUCCESS;
   }
   return FAILURE;
}

/**
 * Deletes the contents of a selected bucket
 *
 * @return Result of operation, SUCCESS if successful, FAILURE if unsuccessful.
 * Process is unsuccessful when the target is not already allocated
 * @param x The x coordinate of the cell to be retrieved
 * @param y The y coordinate of the cell to be retrieved
 */
unsigned int MemoryMonitor::del(unsigned int x, unsigned int y)
{
   unsigned int bit, i, j, k;

   if (references[x*getDimension()+y].firstByte == NULL)
      return FAILURE;

   k = 0;
   for (i = 0; i < getDimension(); i++)
   {
      for (j = 0; j < getDimension(); j++)
      {
         if (references[i*getDimension()+j].firstByte > references[x*getDimension()+y].firstByte && references[i*getDimension()+j].size == references[i*getDimension()+y].size)
            k = i*getDimension()+j;
      }
   }

   hipMemcpy(references[x*getDimension()+y].firstByte, references[k].firstByte, references[k].size, hipMemcpyDeviceToDevice);
   bit = references[k].firstByte - getMemory();
   bitmap->clearBits(bit, references[k].size);

   references[x*getDimension()+y].firstByte = NULL;
   references[x*getDimension()+y].size = 0;

   return SUCCESS;
}

/**
 * Free any dynamic memory assigned to the manager
 */
void MemoryMonitor::cleanup()
{
   delete[] references;
   delete bitmap;
}
